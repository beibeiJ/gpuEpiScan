/* The example is adapted from
https://github.com/sol-prog/cuda_cublas_curand_thrust */


#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "gpuMatrixMultiply.h"


/* Multiply the matrix A and matrix B on GPU and save the result into C */
/* C(m, n) = alpha * A(m, k) %*% B(k, n) + beta * C(m, n) */
/* If it is the matrix product of just A and B, the actual calculation would be
alpha = 1 and beta = 0, so C(m, n) = 1 * A(m, k) % *% B(k, n) + 0 * C(m, n) */

/* *A is the pointer which points to a matrix, same for *B and *C */

void gpu_blas_mm(double *A, double *B, double *C, int m, int k, int n)
{
	int nra = m, nrb = k, nrc = m; /* nr means number of rows */
	const double alf = 1.0;
	const double bet = 0.0;
	const double *alpha = &alf;
	const double *beta = &bet;

	/* create a product for cuBLAS */
	hipblasHandle_t product;
	hipblasCreate(&product);

	/* do the actual multiplication */
	hipblasDgemm(product, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, nra, B, nrb, beta, C, nrc);

	/* destroy the product */
	hipblasDestroy(product);

}


/* gpumm: this function performs matrix multiplication on GPU and send the result back to CPU */
/* C(m, n) = alpha * A(m, k) %*% B(k, n) + beta * C(m, n) */
void gpumm(double *A, double *B, int *m, int *k, int *n, double *C, int *idx)
{
  /* set working GPU(device) */
  hipSetDevice(*idx);

	/* allocate 3 arrays on GPU(device) */
	double *gpuA, *gpuB, *gpuC;
	hipMalloc(&gpuA, *m * *k * sizeof(double)); /* *m and *k is the value while m and k are the address */
	hipMalloc(&gpuB, *k * *n * sizeof(double));
	hipMalloc(&gpuC, *m * *n * sizeof(double));

	/* Copy CPU data to GPU */
	hipMemcpy(gpuA, A, *m * *k * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(gpuB, B, *k * *n * sizeof(double), hipMemcpyHostToDevice);


	/* matrix multiplication on GPU */
	gpu_blas_mm(gpuA, gpuB, gpuC, *m, *k, *n);

	/* copy the result from device to host memory */
	hipMemcpy(C, gpuC, *m * *n * sizeof(double), hipMemcpyDeviceToHost);

	/* free GPU memory */
	hipFree(gpuA);
	hipFree(gpuB);
	hipFree(gpuC);
}


